#include "hip/hip_runtime.h"

#define count 8
#include <iostream>
#include <numeric>
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	auto step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
		}

		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}

int main()
{
	const int size = count * sizeof(int);
	int h[count];
  for(int i=0;i<count;i++)
  {
      h[i] = rand()%100;
  }

	int* d;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count / 2 >>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Sum is " << result << endl;


	return 0;
}