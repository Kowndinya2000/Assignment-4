#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2 // Number of rows of first matrix
#define col1 3 // Number of columns of first matrix
#define row2 3 // Number of rows of second matrix
#define col2 2 // Number of columns of second matrix

__global__ void matproduct(int *a,int *b, int *c)
{
    int row = blockIdx.y*row1+threadIdx.y;
    int col = blockIdx.x*col2+threadIdx.x;
    //printf("%d,%d\n",row,col);
    if(row < row1 && col < col2)
    {
        int val = 0;
        for(int k=0;k<row2;k++)
        {
           val += a[row*col1+k]*b[k*col2+col];
        }
        c[row*col2+col] = val;
    }
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d_a,*d_b,*d_c;

    printf("elements of the first matrix: \n");
    for(int i=0;i<row1;i++)
    {
        for(int j=0;j<col1;j++)
        {
            a[i][j] = rand()%10;
            printf("%d ",a[i][j]);
        }
        printf("\n");
    }
    printf("elements of the second matrix: \n");
    for(int i=0;i<row2;i++)
    {
        for(int j=0;j<col2;j++)
        {
            b[i][j] = rand()%10;
            printf("%d ",b[i][j]);
        }
        printf("\n");
    }

    hipMalloc((void **)&d_a,row1*col1*sizeof(int));
    hipMalloc((void **)&d_b,row2*col2*sizeof(int));
    hipMalloc((void **)&d_c,row1*col2*sizeof(int));

    hipMemcpy(d_a,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);


    dim3 grid((col2+31)/32,(row1+31)/32,1);
    dim3 block(32,32,1);

    matproduct<<<grid,block>>>(d_a,d_b,d_c);
    hipDeviceSynchronize();
    hipMemcpy(c,d_c,row1*col2*sizeof(int),hipMemcpyDeviceToHost);
    printf("Product of two matrices:\n");
    for(int i=0;i<row1;i++)
    {
        for(int j=0;j<col2;j++)
        {
              printf("%d ",c[i][j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}