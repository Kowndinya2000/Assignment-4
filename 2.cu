#include "hip/hip_runtime.h"


#include <stdio.h>
#define arraySize 5

__global__ void addKernel(int* c, const int* a, const int* b, int size) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void addWithCuda(int* c, const int* a, const int* b, int size) {
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<2, (size + 1) / 2>>>(dev_c, dev_a, dev_b, size);
    
    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

int main(int argc, char** argv) {
    
    
    int a[arraySize];
    int b[arraySize];
    for(int i=0; i< arraySize; i++)
    {
        a[i] = rand()%100;
        b[i] = rand()%100;
    }
    int c[arraySize] = { 0 };

    addWithCuda(c, a, b, arraySize);

    for(int i=0; i< arraySize; i++)
    {
        printf("%d ", c[i]);
    }
    hipDeviceReset();

    return 0;
}